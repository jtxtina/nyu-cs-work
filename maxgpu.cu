#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 1024

unsigned int getmax(unsigned int *, unsigned int);
unsigned int getmaxcu(unsigned int *, unsigned int);

void printDeviceInfo()
{
    int nDevices;
    hipGetDeviceCount(&nDevices);

    for (int i = 0; i < nDevices; i++)
    {
        struct hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
               prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
               2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }
} // End of the print device info function




__global__ void getmaxCUDA1(unsigned int * num_d, int new_size, unsigned int * block_result) {
  __shared__ unsigned int local_num[THREADS_PER_BLOCK];
  int tid = threadIdx.x;
  int thread_id = blockDim.x * blockIdx.x + threadIdx.x;

  //Grab the array from the global memory to shared memory.(In one block)
   local_num[tid] = 0;
   if(thread_id < new_size) {
    local_num[tid] = num_d[thread_id];
   }
   __syncthreads();

  for(int i = (blockDim.x)/2; i >= 1 ; i = i / 2) {
    int stride = i;
    if(tid < stride && (tid + stride) < blockDim.x) {
      if(local_num[tid] < local_num[tid+stride]) {
        local_num[tid] = local_num[tid+stride];
      }
    }

    __syncthreads();
  }

  //Get the block max.
  if(tid == 0) {
    block_result[blockIdx.x] = local_num[0];
  }

}



int main(int argc, char *argv[])
{
    unsigned int size = 0;  // The size of the array
    unsigned int i;  // loop index
    unsigned int * numbers; //pointer to the array

    if(argc !=2)
    {
       printf("usage: maxseq num\n");
       printf("num = size of the array\n");
       exit(1);
    }

    size = atol(argv[1]);

    numbers = (unsigned int *)malloc(size * sizeof(unsigned int));
    if( !numbers )
    {
       printf("Unable to allocate mem for an array of size %u\n", size);
       exit(1);
    }

    printDeviceInfo();


   srand(time(NULL)); // setting a seed for the random number generator
    // Fill-up the array with random numbers from 0 to size-1
    for( i = 0; i < size; i++){
       numbers[i] = rand()  % size;
//       printf("number[%d] is %d\n",i, numbers[i]);
    }
    printf(" The maximum number in the array is: %u\n",
           getmaxcu(numbers, size));
    printf(" The maximum number by sequential  in the array is: %u\n",
           getmax(numbers, size));
    free(numbers);
    exit(0);
}


/*
   input: pointer to an array of long int
          number of elements in the array
   output: the maximum number of the array
*/
unsigned int getmaxcu(unsigned int num[], unsigned int size)
{
  unsigned int max = num[0];
 // printf("max[5] is %d max [6] is %d max[1] is %d\n",num[5], num[6], num[1]);

  unsigned int* num_d;
  unsigned int* block_result;
  unsigned int* result;
  unsigned int block_num = size / THREADS_PER_BLOCK;
  if(size % THREADS_PER_BLOCK != 0) {
    block_num = block_num + 1;
  }
  unsigned int block_count = block_num;
  int sizen = size*sizeof(unsigned int);
  int sizeb = block_num*sizeof(unsigned int);
  int new_size = size;

  //1.Transfer num[] to device memory.
  hipMalloc((void**)&num_d, sizen);
  hipMemcpy(num_d, num, sizen, hipMemcpyHostToDevice);

  hipMalloc((void**)&block_result, sizeb);

  //2.Kernel invocation code.
  while(block_num >= 1) {
    if(block_num == 1) {
      getmaxCUDA1<<<block_num, THREADS_PER_BLOCK>>>(num_d, new_size, block_result);
      block_num--;
    } else {
      getmaxCUDA1<<<block_num, THREADS_PER_BLOCK>>>(num_d, new_size, block_result);
      new_size = block_num;
      if(new_size % THREADS_PER_BLOCK != 0) {
        block_num = new_size / THREADS_PER_BLOCK + 1;
      } else {block_num = new_size / THREADS_PER_BLOCK;}
    }
    num_d = block_result;
  }
  result = (unsigned int*) malloc(block_count * sizeof(unsigned int));
  hipMemcpy(result, block_result, sizeb, hipMemcpyDeviceToHost);


  max = result[0];
  //3. Free device memory for num[].
  hipFree(num_d);
  hipFree(block_result);


  return( max );

}

unsigned int getmax(unsigned int num[], unsigned int size)
{
  unsigned int i;
  unsigned int max = num[0];

  for(i = 1; i < size; i++)
	if(num[i] > max)
	   max = num[i];

  return( max );

}
